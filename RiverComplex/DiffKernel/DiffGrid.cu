#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>

extern "C" {
	// Device code
	__global__ void DiffGrid(float* X, float* Y, int* N)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int Nx = N[0];
		int Ny = N[1];		
		
		if ((i > Nx) & (i < Nx * Ny - Nx))
			if ((i % Nx != 0) & ((i + 1) % Nx != 0))
			{
				X[i] = 0.25 * (X[i + 1] + X[i - 1] + X[i - Nx] + X[i + Nx]);
				Y[i] = 0.25 * (Y[i + 1] + Y[i - 1] + Y[i - Nx] + Y[i + Nx]);
			}
	}
}