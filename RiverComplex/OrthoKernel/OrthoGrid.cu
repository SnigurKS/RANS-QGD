#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>

extern "C" {
	// Device code
	__global__ void OrthoGrid(float* X, float* Y, float* ar, int* N)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		float RelaxOrto = ar[0];
		float Tay = ar[1];
		int Nx = N[0];
		int Ny = N[1];
		
				
		if ((i > Nx) & (i < Nx * Ny - Nx))
			if ((i % Nx != 0) & ((i + 1) % Nx != 0))
			{
				float xp = 0; float xe = 0; float xw = 0; float xs = 0; float xn = 0;
				float yp = 0; float ye = 0; float yw = 0; float ys = 0; float yn = 0;
				float xen = 0; float xwn = 0; float xes = 0; float xws = 0;
				float yen = 0; float ywn = 0; float yes = 0; float yws = 0;
				float Ap = 0; float Ig = 0; float Alpha = 0; float Betta = 0; float Gamma = 0; float Delta = 0;

				xp = X[i];
				xe = X[i + 1];
				xw = X[i - 1];
				xs = X[i - Nx];
				xes = X[i - Nx + 1];
				xws = X[i - Nx - 1];

				xn = X[i + Nx];
				xen = X[i + Nx + 1];
				xwn = X[i + Nx - 1];

				yp = Y[i];
				ye = Y[i + 1];
				yw = Y[i - 1];
				ys = Y[i - Nx];
				yes = Y[i - Nx + 1];
				yws = Y[i - Nx - 1];

				yn = Y[i + Nx];
				yen = Y[i + Nx + 1];
				ywn = Y[i + Nx - 1];

				/// g22
				Alpha = 0.25 * ((xn - xs) * (xn - xs) + (yn - ys) * (yn - ys));
				/// g12
				Betta = RelaxOrto * 0.25 * ((xe - xw) * (xn - xs) + (ye - yw) * (yn - ys));
				/// g11
				Gamma = 0.25 * ((xe - xw) * (xe - xw) + (ye - yw) * (ye - yw));
				/// ����� �� �������� �� float
				if ((Alpha + Gamma) < 0.000001)
				{
					Alpha = 1;
					Gamma = 1;
					Betta = 0;
				}
				//
				Ig = Alpha + Gamma;
				Ap = 1.0 / (2 * Ig);

				xp = Ap * (Alpha * (xw + xe) + Gamma * (xn + xs) - 0.5 * Betta * (xen - xwn - xes + xws));

				yp = Ap * (Alpha * (yw + ye) + Gamma * (yn + ys) - 0.5 * Betta * (yen - ywn - yes + yws));

				X[i] = (1 - Tay) * X[i] + Tay * xp;
				Y[i] = (1 - Tay) * Y[i] + Tay * yp;
			} //*/
	}
}