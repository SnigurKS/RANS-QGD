#include "hip/hip_runtime.h"
/*
* This code is taken more or less entirely from the NVIDIA CUDA SDK.
* This software contains source code provided by NVIDIA Corporation.
*
*/


//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>


extern "C" {
	// Device code
	__global__ void AlgGrid(const double* TopX, const double* BotX, const double* TopY, const double* BotY, double Q, double P, int Nx, int Ny, double* X, double* Y)
	{
		int idx = blockDim.x * blockIdx.x + threadIdx.x;

		int i = idx / Nx; // row number
		int j = idx % Nx; // col number
		// mesh step by vertical
		double DETA = 1.0 / (Ny - 1);
		double TQI = 1.0f / tanhf(Q);
		double ETA = i * DETA;
		double DUM = Q * (1 - ETA);
		DUM = 1 - tanhf(DUM) * TQI;
		double s = P * ETA + (1 - P) * DUM;
		// form functions
		double N0 = s;
		double N1 = 1 -s;

		if (idx < Nx * Ny) {
			X[idx] = N0*BotX[j] + N1*TopX[j];
			Y[idx] = N0*BotY[j] + N1*TopY[j];
		}
	}
}
/*
* This code is taken more or less entirely from the NVIDIA CUDA SDK.
* This software contains source code provided by NVIDIA Corporation.
*
*/


