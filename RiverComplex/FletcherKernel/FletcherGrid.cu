#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>

extern "C" {
	// Device code
	__global__ void Surch(int* jMax, float* SS2, float* XS, float* YS)
	{
		int j = blockDim.x * blockIdx.x + threadIdx.x;
		int Nx = jMax[0];
		if (j >= Nx) return;
		float S2 = SS2[0];
		float S3 = SS2[1];
		int jMap = Nx - 1;
		float EM1, EM2, EM3, EM4;
		float X2, Y2, X3, Y3;
		float STJM, SJJM;
		float XS2;
		float YS2;
		float XS3;
		float YS3;
		//
		float DXS = XS[3 * Nx + j] - XS[0 * Nx + j];
		float DYS = YS[3 * Nx + j] - YS[0 * Nx + j];
		XS[1 * Nx + j] = XS[0 * Nx + j] + S2 * DXS;
		YS[1 * Nx + j] = YS[0 * Nx + j] + S2 * DYS;
		XS[2 * Nx + j] = XS[0 * Nx + j] + S3 * DXS;
		YS[2 * Nx + j] = YS[0 * Nx + j] + S3 * DYS;
		//
		if ((j>1)&(j<jMap))
		{
			if (fabsf(XS[0 * Nx + j + 1] - XS[0 * Nx + j - 1]) > 0.000001)
				EM1 = (YS[0 * Nx + j + 1] - YS[0 * Nx + j - 1]) / (XS[0 * Nx + j + 1] - XS[0 * Nx + j - 1]);
			else
				EM1 = 1.0E+06f * (YS[0 * Nx + j + 1] - YS[0 * Nx + j - 1]);
			if (fabsf(XS[1 * Nx + j] - XS[1 * Nx + j - 1]) > 0.000001)
				EM2 = (YS[1 * Nx + j] - YS[1 * Nx + j - 1]) / (XS[1 * Nx + j] - XS[1 * Nx + j - 1]);
			else
				EM2 = 1000000 * (YS[1 * Nx + j] - YS[1 * Nx + j - 1]);
			X2 = (EM1 * (YS[0 * Nx + j] - YS[1 * Nx + j] + EM2 * XS[1 * Nx + j]) + XS[0 * Nx + j]) / (1 + EM1 * EM2);
			Y2 = YS[1 * Nx + j] + EM2 * (X2 - XS[1 * Nx + j]);
			STJM = sqrtf((X2 - XS[1 * Nx + j - 1]) * (X2 - XS[1 * Nx + j - 1]) + (Y2 - YS[1 * Nx + j - 1]) * (Y2 - YS[1 * Nx + j - 1]));
			SJJM = sqrtf((XS[1 * Nx + j] - XS[1 * Nx + j - 1]) * (XS[1 * Nx + j] - XS[1 * Nx + j - 1]) + (YS[1 * Nx + j] - YS[1 * Nx + j - 1]) * (YS[1 * Nx + j] - YS[1 * Nx + j - 1]));
			if (STJM < SJJM)
			{
				XS2 = X2;
				YS2 = Y2;
			}
			else
			{
				if (fabsf(XS[1 * Nx + j + 1] - XS[1 * Nx + j]) > 0.000001)
					EM2 = (YS[1 * Nx + j + 1] - YS[1 * Nx + j]) / (XS[1 * Nx + j + 1] - XS[1 * Nx + j]);
				else
					EM2 = 1000000 * (YS[1 * Nx + j + 1] - YS[1 * Nx + j]);
				X2 = (EM1 * (YS[0 * Nx + j] - YS[1 * Nx + j] + EM2 * XS[1 * Nx + j]) + XS[0 * Nx + j]) / (1 + EM1 * EM2);
				Y2 = YS[1 * Nx + j] + EM2 * (X2 - XS[1 * Nx + j]);
				XS2 = X2;
				YS2 = Y2;
			}

			if (fabsf(XS[3 * Nx + j + 1] - XS[3 * Nx + j - 1]) > 0.000001)
				EM4 = (YS[3 * Nx + j + 1] - YS[3 * Nx + j - 1]) / (XS[3 * Nx + j + 1] - XS[3 * Nx + j - 1]);
			else
				EM4 = 1000000 * (YS[3 * Nx + j + 1] - YS[3 * Nx + j - 1]);
			if (fabsf(XS[2 * Nx + j] - XS[2 * Nx + j - 1]) > 0.000001)
				EM3 = (YS[2 * Nx + j] - YS[2 * Nx + j - 1]) / (XS[2 * Nx + j] - XS[2 * Nx + j - 1]);
			else
				EM3 = 1000000 * (YS[2 * Nx + j] - YS[2 * Nx + j - 1]);
			//
			X3 = (EM4 * (YS[3 * Nx + j] - YS[2 * Nx + j] + EM3 * XS[2 * Nx + j]) + XS[3 * Nx + j]) / (1 + EM3 * EM4);
			Y3 = YS[2 * Nx + j] + EM3 * (X3 - XS[2 * Nx + j]);
			STJM = sqrtf((X3 - XS[2 * Nx + j - 1]) * (X3 - XS[2 * Nx + j - 1]) + (Y3 - YS[2 * Nx + j - 1]) * (Y3 - YS[2 * Nx + j - 1]));
			SJJM = sqrtf((XS[2 * Nx + j] - XS[2 * Nx + j - 1]) * (XS[2 * Nx + j] - XS[2 * Nx + j - 1]) + (YS[2 * Nx + j] - YS[2 * Nx + j - 1]) * (YS[2 * Nx + j] - YS[2 * Nx + j - 1]));
			//
			if (STJM > SJJM)
			{
				if (fabsf(XS[2 * Nx + j + 1] - XS[2 * Nx + j]) > 0.000001)
					EM3 = (YS[2 * Nx + j + 1] - YS[2 * Nx + j]) / (XS[2 * Nx + j + 1] - XS[2 * Nx + j]);
				else
					EM3 = 1000000 * (YS[2 * Nx + j + 1] - YS[2 * Nx + j]);
				X3 = (EM4 * (YS[3 * Nx + j] - YS[2 * Nx + j] + EM3 * XS[2 * Nx + j]) + XS[3 * Nx + j]) / (1 + EM3 * EM4);
				Y3 = YS[2 * Nx + j] + EM3 * (X3 - XS[2 * Nx + j]);
			}
			//
			XS3 = X3;
			YS3 = Y3;

			XS[1 * Nx + j] = XS2;
			YS[1 * Nx + j] = YS2;
			XS[2 * Nx + j] = XS3;
			YS[2 * Nx + j] = YS3;

		}
	}

	__global__ void InternalGrid(float* X, float* Y, float* XS, float* YS, float* sCD, float* sAF, float* Aw, int* N)
	{
		//int width_j = 4;
		int idx = blockDim.x * blockIdx.x + threadIdx.x;
		int Nx = N[0];
		int Ny = N[1];
		int j = idx % Nx;
		int k = idx / Nx;
		//int idj = j * width_j;
		//
		float AW = Aw[0];		
		float A1 = 2.0f / (3.0f * AW - 1);
		float A2 = 2.0f / (3.0f * (1 - AW) - 1);
		float AJM = Nx - 1;
		float DZI = 1.0f / AJM;
		//
		float AJ, ZI, S;
		float sH[4];
		int l;
		
		// ������ if ((idj+1)<Nx)
		// l = fminf(j, Nx - 1);
		if (idx < Nx * Ny) {
			AJ = j - 1;
			ZI = AJ * DZI;
			S = sAF[k] + ZI * (sCD[k] - sAF[k]);
			//
			sH[0] = (1 - S) * (1 - S) * (1 - A1 * S);
			sH[1] = (1 - S) * (1 - S) * S * (A1 + 2);
			sH[2] = (1 - S) * S * S * (A2 + 2);
			sH[3] = S * S * (1 - A2 * (1 - S));
			//�������� ���������
			for (int L = 0; L < 4; L++)
			{
				X[j + k*Nx] = X[j + k*Nx] + sH[L] * XS[L * Nx + j];
				Y[j + k*Nx] = Y[j + k*Nx] + sH[L] * YS[L * Nx + j];
			}
		}	
		
	}
}