#include "hip/hip_runtime.h"
/*
* This code is taken more or less entirely from the NVIDIA CUDA SDK.
* This software contains source code provided by NVIDIA Corporation.
*
*/


//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>


extern "C" {
	// Device code
	__global__ void UVKernel(const int width, const float dt, const float rhow, const float nu, const float tau, const int CVLength, int* OCV, int* Num, float* OLx10, float* OLx32, float* OLy01, float* OLy23, float* OSS, int* OP1, int* OAreaElems, float* ONx, float* ONy, float* OAlpha, float* OLk, float* OS0, float* OP, float* OU, float* OV)
	{
		int c = (blockDim.x * blockIdx.x + threadIdx.x) * width; // NX*NY / width
		if (c < CVLength)
		{
			//
			float LsummU = 0, LsummV = 0; //������ U, V ��������
			int p0, jj, Lv1, Lv2, Lp1, Lt1, Lt2, Lt3, Lz1, Lz2, Lz3;
			int Knots1[3];
			int Knots2[3];
			float lx10, lx32, ly01, ly23, LS, LUc1, LVc1, LPc1, LUc2, LVc2, LPc2, Ls2, Ldudx, Ldudy, Ldvdx, Ldvdy, Ldpdx, Ldpdy, Lnx, Lny, Lalpha, LUcr, LVcr, LPcr, LLk, Lpress, LconvU, LdiffU,
				LregU1, LregU2, LregU, LpressV, LconvV, LdiffV, LregV1, LregV2, LregV;
			int k, i, j;
			for (k = 0; k < width; k++)
			{
				i = c + k;
				//
				LsummU = 0; //������ U ��������
				LsummV = 0; //������ V ��������
				//
				p0 = OCV[Num[i]];
				jj = Num[i + 1] - Num[i] - 1; //���������� ��, ��������� � ������ �����
				//
				for (j = Num[i]; j < Num[i + 1] - 1; j++)
				{
					lx10 = OLx10[j]; lx32 = OLx32[j];
					ly01 = OLy01[j]; ly23 = OLy23[j];
					//�������
					LS = OSS[j];
					//��������� ��������
					Lv1 = OCV[(j - Num[i] + 1) % jj + Num[i] + 1];
					Lv2 = OCV[j + 1];
					//������ ����� ����� �����
					Lp1 = OP1[j];
					//������� �������� ������� � ������� ���� 1��� � 2��� ������������ ��� ������� �������� �� ��������
					Knots1[0] = OAreaElems[Lv1 * 3]; Knots1[1] = OAreaElems[Lv1 * 3 + 1]; Knots1[2] = OAreaElems[Lv1 * 3 + 2];
					Lt1 = Knots1[0]; Lt2 = Knots1[1]; Lt3 = Knots1[2];
					LUc1 = (OU[Lt1] + OU[Lt2] + OU[Lt3]) / 3.0;
					LVc1 = (OV[Lt1] + OV[Lt2] + OV[Lt3]) / 3.0;
					LPc1 = (OP[Lt1] + OP[Lt2] + OP[Lt3]) / 3.0;
					Knots2[0] = OAreaElems[Lv2 * 3]; Knots2[1] = OAreaElems[Lv2 * 3 + 1]; Knots2[2] = OAreaElems[Lv2 * 3 + 2];
					Lz1 = Knots2[0]; Lz2 = Knots2[1]; Lz3 = Knots2[2];
					LUc2 = (OU[Lz1] + OU[Lz2] + OU[Lz3]) / 3.0;
					LVc2 = (OV[Lz1] + OV[Lz2] + OV[Lz3]) / 3.0;
					LPc2 = (OP[Lz1] + OP[Lz2] + OP[Lz3]) / 3.0;
					//�������� ����������� � ����� ����������� ������
					Ls2 = 2 * LS;
					Ldudx = ((LUc1 - LUc2) * ly01 + (OU[Lp1] - OU[p0]) * ly23) / Ls2;
					Ldudy = ((LUc1 - LUc2) * lx10 + (OU[Lp1] - OU[p0]) * lx32) / Ls2;
					Ldvdx = ((LVc1 - LVc2) * ly01 + (OV[Lp1] - OV[p0]) * ly23) / Ls2;
					Ldvdy = ((LVc1 - LVc2) * lx10 + (OV[Lp1] - OV[p0]) * lx32) / Ls2;
					Ldpdx = ((LPc1 - LPc2) * ly01 + (OP[Lp1] - OP[p0]) * ly23) / Ls2;
					Ldpdy = ((LPc1 - LPc2) * lx10 + (OP[Lp1] - OP[p0]) * lx32) / Ls2;
					//������� ������� � ����� �� (������� ��)
					Lnx = ONx[j]; Lny = ONy[j];
					////�������� ������� � ����� ����������� ����� �� � �������� �����
					Lalpha = OAlpha[j];
					LUcr = Lalpha * OU[p0] + (1 - Lalpha) * OU[Lp1];
					LVcr = Lalpha * OV[p0] + (1 - Lalpha) * OV[Lp1];
					LPcr = Lalpha * OP[p0] + (1 - Lalpha) * OP[Lp1];
					//����� �������� ��������� �������� ������� ��
					LLk = OLk[j];
					//������ �������
					Lpress = -1.0 / rhow * LPcr * Lnx;
					LconvU = -LUcr * LUcr * Lnx - (LUcr * LVcr) * Lny;
					LdiffU = nu * (2.0 * Ldudx * Lnx - 2.0 / 3.0 * (Ldudx + Ldvdy) * Lnx + Ldudy * Lny + Ldvdx * Lny);
					LregU1 = 2.0 * tau * LUcr * (LUcr * Ldudx + LVcr * Ldudy + 1.0 / rhow * Ldpdx) * Lnx;
					LregU2 = tau * (LVcr * (LUcr * Ldudx + LVcr * Ldudy + 1.0 / rhow * Ldpdx) + LUcr * (LUcr * Ldvdx + LVcr * Ldvdy + 1.0 / rhow * Ldpdy)) * Lny;
					LregU = LregU1 + LregU2;
					LsummU += (LconvU + LdiffU + LregU + Lpress) * LLk;
					//                  
					LpressV = -1.0 / rhow * LPcr * Lny;
					LconvV = -(LUcr * LVcr) * Lnx - LVcr * LVcr * Lny;
					LdiffV = nu * (2.0 * Ldvdy * Lny - 2.0 / 3.0 * (Ldudx + Ldvdy) * Lny + Ldvdx * Lnx + Ldudy * Lnx);
					LregV1 = 2.0 * tau * LVcr * (LUcr * Ldvdx + LVcr * Ldvdy + 1.0 / rhow * Ldpdy) * Lny;
					LregV2 = tau * (LVcr * (LUcr * Ldudx + LVcr * Ldudy + 1.0 / rhow * Ldpdx) + LUcr * (LUcr * Ldvdx + LVcr * Ldvdy + 1.0 / rhow * Ldpdy)) * Lnx;
					LregV = LregV1 + LregV2;
					LsummV += (LconvV + LdiffV + LregV + LpressV) * LLk;
				}
				//

				OU[p0] = OU[p0] + (dt / OS0[i] * LsummU);
				OV[p0] = OV[p0] + (dt / OS0[i] * LsummV);
				//


			}
		}
	}
}